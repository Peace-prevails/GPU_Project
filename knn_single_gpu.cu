#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include<time.h>
#include <float.h>

#define NUM_FEATURES 4
#define MAX_DATA_SIZE 30001
#define NUM_CLASSES 3
#define TRAIN_TEST_SPLIT 0.8
double train_features[MAX_DATA_SIZE * NUM_FEATURES];
int train_labels[MAX_DATA_SIZE];
double test_features[MAX_DATA_SIZE * NUM_FEATURES];
int test_labels[MAX_DATA_SIZE];
// int labelToInt(const char* label) {
//     if (strcmp(label, "Setosa") == 0) return 0;
//     if (strcmp(label, "Virginica") == 0) return 1;
//     if (strcmp(label, "Versicolor") == 0) return 2;
//     printf("Unrecognized label: %s\n", label);
//     return -1;
// }
int labelToInt(const char* label) {
    if (strcmp(label, "low") == 0) return 0;
    if (strcmp(label, "medium") == 0) return 1;
    if (strcmp(label, "high") == 0) return 2;
    printf("Unrecognized label: %s\n", label);
    return -1;
}

int loadData(const char* filename, double* features, int* labels) {
    FILE* file = fopen(filename, "r");
    if (file == NULL) {
        perror("Error opening file");
        return -1;
    }

    char line[1024];
    int count = 0;
    fgets(line, sizeof(line), file); // 跳过标题行

    while (fgets(line, sizeof(line), file) && count < MAX_DATA_SIZE) {
        char* token = strtok(line, ",");
        for (int i = 0; i < NUM_FEATURES; i++) {
            if (token == NULL) {
                fprintf(stderr, "Error in data format\n");
                fclose(file);
                return -1;
            }
            features[count * NUM_FEATURES + i] = atof(token);
            token = strtok(NULL, ",");
        }

        if (token == NULL) {
            fprintf(stderr, "Error in data format\n");
            fclose(file);
            return -1;
        }
        token[strcspn(token, "\n")] = 0; // 移除换行符
        labels[count] = labelToInt(token);
        count++;
    }

    fclose(file);
    return count;
}

__global__ void computeDistances(double *train_features, double *test_features, double *distances, int train_size, int test_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < test_size) {
        for (int i = 0; i < train_size; i++) {
            double dist = 0.0;
            for (int j = 0; j < NUM_FEATURES; j++) {
                double diff = train_features[i * NUM_FEATURES + j] - test_features[idx * NUM_FEATURES + j];
                dist += diff * diff;
            }
            
            distances[idx * train_size + i] = sqrt(dist);
        }
    }
}

int majorityVote(double *distances, int *train_labels, int train_size, int idx, int k) {
    int votes[NUM_CLASSES] = {0};
    double temp_distances[train_size];

    // 复制对应测试样本的距离
    for (int i = 0; i < train_size; i++) {
        temp_distances[i] = distances[idx * train_size + i];
    }

    for (int i = 0; i < k; i++) {
        double min_dist = DBL_MAX;
        int min_index = -1;
        for (int j = 0; j < train_size; j++) {
            if (temp_distances[j] < min_dist) {
                min_dist = temp_distances[j];
                min_index = j;
            }
        }

        if (min_index != -1) {
            votes[train_labels[min_index]]++;
            temp_distances[min_index] = DBL_MAX; // 将已选择的最小距离设置为最大值，以便在下一轮中忽略它
        }
    }

    int max_votes = 0, predicted_label = -1;
    for (int i = 0; i < NUM_CLASSES; i++) {
        if (votes[i] > max_votes) {
            max_votes = votes[i];
            predicted_label = i;
        }
    }
    return predicted_label;
}


int main() {
    // Load data into flat arrays
    int total_size = loadData("synthetic_knn_dataset.csv", train_features, train_labels);
    if (total_size == -1) return 1;
    
    // printf("First 5 samples of train_features:\n");
    // for (int i = 0; i < 5; ++i) {
    //     for (int j = 0; j < NUM_FEATURES; ++j) {
    //         printf("%f ", train_features[i * NUM_FEATURES + j]);
    //     }
    //     printf("\n");
    // }

    // // 打印前5个样本的标签
    // printf("First 5 labels of train_labels:\n");
    // for (int i = 0; i < 5; ++i) {
    //     printf("%d\n", train_labels[i]);
    // }

    int train_size = (int)(total_size * TRAIN_TEST_SPLIT);
    int test_size = total_size - train_size;
    for (int i = 0; i < test_size; i++) {
        for (int j = 0; j < NUM_FEATURES; j++) {
            test_features[i * NUM_FEATURES + j] = train_features[(train_size + i) * NUM_FEATURES + j];
        }
        test_labels[i] = train_labels[train_size + i];
    }
    double time_taken2;
    double time_taken1,time_taken3;
    clock_t start, end;
    start = clock();
    // Allocate memory on GPU
    
    double *d_train_features, *d_test_features, *d_distances;
    hipMalloc(&d_train_features, train_size * NUM_FEATURES * sizeof(double));
    hipMalloc(&d_test_features, test_size * NUM_FEATURES * sizeof(double));
    hipMalloc(&d_distances, train_size * test_size * sizeof(double));

    // Copy training and testing data to GPU
    hipMemcpy(d_train_features, train_features, train_size * NUM_FEATURES * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_test_features, test_features, test_size * NUM_FEATURES * sizeof(double), hipMemcpyHostToDevice);

    // Setup kernel execution parameters and launch kernel
    
    dim3 blockSize(128);
    dim3 gridSize((test_size + blockSize.x - 1) / blockSize.x);
    computeDistances<<<gridSize, blockSize>>>(d_train_features, d_test_features, d_distances, train_size, test_size);
    end = clock();
    hipDeviceSynchronize();
    time_taken1 = ((double)(end - start))/ CLOCKS_PER_SEC;
    // Allocate memory on host for the distances and copy from device to host
    double *h_distances = (double*)malloc(train_size * test_size * sizeof(double));
    start = clock();
    hipMemcpy(h_distances, d_distances, train_size * test_size * sizeof(double), hipMemcpyDeviceToHost);
    end = clock();
    time_taken2 = ((double)(end - start))/ CLOCKS_PER_SEC;
    
    // Here is the logic for the k nearest neighbors and majority voting
    double best_accuracy = 0.0;
    int best_k = 1;
    start=clock();
    // 尝试不同的k值
    for (int k = 1; k <= 5; k++) {
        int correct_predictions = 0;

        // 对于测试集中的每个点
        for (int i = 0; i < test_size; i++) {
            int predicted_label = majorityVote(h_distances, train_labels, train_size, i, k);
            if (predicted_label == test_labels[i]) {
                correct_predictions++;
            }
        }

        double accuracy = (double)correct_predictions / test_size;
        // printf("k = %d, Accuracy: %f\n", k, accuracy);

        // 更新最佳k值
        if (accuracy > best_accuracy) {
            best_accuracy = accuracy;
            best_k = k;
        }
    }
   end=clock();
   time_taken3 = ((double)(end - start))/ CLOCKS_PER_SEC;
   printf("Time taken for copying data to device,launch kernels: %lf\n", time_taken1);
   printf("Time taken for copying data back to the host: %lf\n", time_taken2);
   printf("Total overhead: %lf\n", time_taken1+time_taken2);
   printf("Total time: %lf\n", time_taken1+time_taken2+time_taken3);
   printf("Best k value: %d with accuracy: %f\n", best_k, best_accuracy);


    // Free GPU memory
    hipFree(d_train_features);
    hipFree(d_test_features);
    hipFree(d_distances);

    // Free host memory
    // free(h_distances);

    return 0;
}
