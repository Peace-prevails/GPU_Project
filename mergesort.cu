
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>


__device__ void gpuMerge(int* source, int* dest, int start, int mid, int end) {
    int i = start;
    int j = mid;
    for (int k = start; k < end; k++) {
        if (i < mid && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

__global__ void mergeSortKernel(int* source, int* dest, int width, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int stride = blockDim.x * gridDim.x * width;

    for (int start = index * width; start < len; start += stride) {
      int mid = min(start + (width >> 1), len);
      int end = min(start + width, len);
      gpuMerge(source, dest, start, mid, end);
    }

}

void mergeSortSingleGPU(int* list, int len) {
    int* d_list, *d_temp;
    int blocks = 1024;
    int threadsPerBlock = 256;

    double time_malloc = 0, time_memcpy = 0, time_launch = 0, time_run = 0;
    clock_t start, end;

    // Allocate and copy device memory
    start = clock();
    hipMalloc(&d_list, len * sizeof(int));
    hipMalloc(&d_temp, len * sizeof(int));
    end = clock();
    time_malloc += ((double)(end-start)) / CLOCKS_PER_SEC;

    start = clock();
    hipMemcpy(d_list, list, len * sizeof(int), hipMemcpyHostToDevice);
    end = clock();
    time_memcpy += ((double)(end-start)) / CLOCKS_PER_SEC;
    // Launch kernel

    for (int width = 2; width < (len << 1); width <<= 1) {
      start = clock();
      mergeSortKernel<<<blocks, threadsPerBlock>>>(d_list, d_temp, width, len);
      end = clock();
      time_launch += ((double)(end-start)) / CLOCKS_PER_SEC;

      hipDeviceSynchronize();
      end = clock();
      time_run += ((double)(end-start)) / CLOCKS_PER_SEC;

      int* temp = d_list;
      d_list = d_temp;
      d_temp = temp;
    }
    // Copy result back to host
    start = clock();
    hipMemcpy(list, d_list, len * sizeof(int), hipMemcpyDeviceToHost);
    end = clock();
    time_memcpy += ((double)(end-start)) / CLOCKS_PER_SEC;

    hipFree(d_list);
    hipFree(d_temp);
    printf("hipMalloc time = %lf secs\n", time_malloc); 
    printf("hipMemcpy time = %lf secs\n", time_memcpy); 
    printf("kernel launch time = %lf secs\n", time_launch); 
    printf("kernel run time = %lf secs\n", time_run); 
}

void mergeSortMulGPU(int* list, int len) {
    int *d_list, *d_temp, *d_list1, *d_temp1, *d_list2, *d_temp2;
    int blocks = 1024;
    int threadsPerBlock = 256;

    // Allocate and copy device memory
    
    //cudaMemcpy(d_list, list, len * sizeof(int), cudaMemcpyHostToDevice);
    
    hipSetDevice(0);
    hipMalloc(&d_list, len * sizeof(int));
    hipMalloc(&d_temp, len * sizeof(int));


    hipMalloc(&d_list1, len/2 * sizeof(int));
    hipMalloc(&d_temp1, len/2 * sizeof(int));
    hipMemcpy(d_list1, list, len / 2 * sizeof(int), hipMemcpyHostToDevice);

    hipSetDevice(1);
    hipMalloc(&d_list2, len/2 * sizeof(int));
    hipMalloc(&d_temp2, len/2 * sizeof(int));
    hipMemcpy(d_list2, &list[len/2], len / 2 * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel 
    for (int width = 2; width < (len << 1); width <<= 1) {
      hipSetDevice(0);
      mergeSortKernel<<<blocks, threadsPerBlock>>>(d_list1, d_temp1, width, len/2);
      hipSetDevice(1);
      mergeSortKernel<<<blocks, threadsPerBlock>>>(d_list2, d_temp2, width, len/2);

      hipSetDevice(0);
      hipDeviceSynchronize();
      hipSetDevice(1);
      hipDeviceSynchronize();

      int* temp = d_list1;
      d_list1 = d_temp1;
      d_temp1 = temp;

      temp = d_list2;
      d_list2 = d_temp2;
      d_temp2 = temp;
    }
    
    
    // Copy result back to host
    hipSetDevice(0);
    hipMemcpy(list, d_list1, len/2 * sizeof(int), hipMemcpyDeviceToHost);
    hipSetDevice(1);
    hipMemcpy(&list[len/2], d_list2, len/2 * sizeof(int), hipMemcpyDeviceToHost);

    //final merge
    hipSetDevice(0);
    hipMemcpy(d_list, list, len * sizeof(int), hipMemcpyHostToDevice);
    mergeSortKernel<<<blocks, threadsPerBlock>>>(d_list, d_temp, len, len);
    hipMemcpy(list, d_temp, len * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_list);
    hipFree(d_temp);
    hipFree(d_list1);
    hipFree(d_temp1);
    hipFree(d_list2);
    hipFree(d_temp2);
}


int main(int argc, char *argv[]) {
  long int n = 0, type_of_device = 0;

  // to measure the time
  double time_taken = 0;
  clock_t start, end;

  if(argc != 3)
  {
    fprintf(stderr, "usage: mergesort n who\n");
    fprintf(stderr, "n = size of int array to be sorted\n");
    fprintf(stderr, "who = 0: sequential code on CPU, 1: Single GPU version, 2: Multi-GPU version\n");
    exit(1);
  }
  n = atoi(argv[1]);
  type_of_device = atoi(argv[2]);

  // generate random list
  int* list;
  list = (int *) malloc(n * sizeof(int));
  
  srand(time(NULL));
  for (int i = 0; i < n; i++) {
        list[i] = rand(); // Generates a random number
  }

  // printf("Original array:\n");
  // for (int i = 0; i < n; ++i) {
  //   printf("%d\n", list[i]);
  // }

  switch(type_of_device) {
    case 0: 
        printf("Sequential version:\n");

        start = clock();
        std::sort(list, list + n);
        end = clock();  // end of measuring
        time_taken = ((double)(end-start)) / CLOCKS_PER_SEC;
        printf("CPU time = %lf secs\n", time_taken); 
        break;
	case 1: 
        printf("Single GPU version:\n");

        start = clock();
        mergeSortSingleGPU(list, n);
        end = clock();  // end of measuring
        time_taken = ((double)(end-start)) / CLOCKS_PER_SEC;
        printf("Single GPU time = %lf secs\n", time_taken); 
        break;
	case 2: 
        printf("Multi GPU version:\n");

        start = clock();
        mergeSortMulGPU(list, n);
        end = clock();  // end of measuring
        time_taken = ((double)(end-start)) / CLOCKS_PER_SEC;
        printf("Multi GPU time = %lf secs\n", time_taken); 
        break;
	default: 
        printf("Invalid device type\n");
		exit(1);
  }

  // printf("Sorted array:\n");
  // for (int i = 0; i < n; ++i) {
  //   printf("%d\n", list[i]);
  // }
  free(list);
}